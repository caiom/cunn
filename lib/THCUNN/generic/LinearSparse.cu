
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/LinearSparse.cu"
#else

#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256
#define NUM_BLOCKS(n) (int)THCCeilDiv(n, (long) BLOCK_SIZE)

void THNN_(LinearSparse_updateOutput)(
           THCState * state,
           THCTensor * inputT,
           THCTensor * outputT,
           THCTensor * weightT,
           THCudaIntTensor * rowsT,
           THCudaIntTensor * colsT,
           THCudaIntTensor * iRowStart,
           int nRowWeight,
           int nColWeight) {

  THAssert(THCudaTensor_checkGPU(state, 6, inputT, outputT, weightT,
                                 rowsT, colsT, iRowStart));
  //THArgCheck(inputT->nDimension == 2, 4, "Input should be 2D");
  //THArgCheck(outputT->nDimension == 2, 4, "output tensor must be 2D");

  int nRowInput = inputT->size[0];
  int nColInput = inputT->size[1];

  int nnz = weightT->size[0];

  char transa = 'n';
  char transb = 'n';

  real alpha = 1.0;
  real beta = 0.0;

  if(inputT->nDimension == 1)
  {
    beta = 1.0f;

    #ifdef THC_REAL_IS_FLOAT
    THCusparse_Scsrmv(state,
    #elif defined(THC_REAL_IS_DOUBLE)
    THCusparse_Dcsrmv(state,
    #endif
                     transa, nRowWeight, nColWeight, nnz, alpha, THCTensor_(data)(state, weightT), THCudaIntTensor_data(state, iRowStart), 
                     THCudaIntTensor_data(state, colsT), THCTensor_(data)(state, inputT), beta, THCTensor_(data)(state, outputT));
  }
  else
  {

    #ifdef THC_REAL_IS_FLOAT
    THCusparse_Scsrmm2(state,
    #elif defined(THC_REAL_IS_DOUBLE)
    THCusparse_Dcsrmm2(state,
    #endif
                      transa, transb, nRowWeight, nRowInput, nColWeight, nnz, alpha, 
                      THCTensor_(data)(state, weightT), THCudaIntTensor_data(state, iRowStart), 
                      THCudaIntTensor_data(state, colsT), THCTensor_(data)(state, inputT), nColInput, beta, 
                      THCTensor_(data)(state, outputT), nRowWeight);
  }
}

void THNN_(LinearSparse_updateGradInput)(
           THCState * state,
           THCTensor * inputDummy,
           THCTensor * inputT,
           THCTensor * outputT,
           THCTensor * weightT,
           THCudaIntTensor * rowsT,
           THCudaIntTensor * colsT,
           int nRowWeight,
           int nColWeight,
           THCudaIntTensor * iRowStartT) {

  THAssert(THCudaTensor_checkGPU(state, 6, inputT, outputT, weightT,
                                 rowsT, colsT, iRowStartT));

  //int nRowInput = inputT->size[0];
  int nColInput = inputT->size[1];
  int nRowOutput = outputT->size[0];
  int nColOutput = outputT->size[1];

  int nnz = weightT->size[0];

  char transa = 't';
  char transb = 'n';

  real alpha = 1.0;
  real beta = 0.0;

  if(inputT->nDimension == 1)
  {

    #ifdef THC_REAL_IS_FLOAT
    THCusparse_Scsrmv(state,
    #elif defined(THC_REAL_IS_DOUBLE)
    THCusparse_Dcsrmv(state,
    #endif
                     transa, nRowWeight, nColWeight, nnz, alpha, THCTensor_(data)(state, weightT), THCudaIntTensor_data(state, iRowStartT), 
                     THCudaIntTensor_data(state, colsT), THCTensor_(data)(state, inputT), beta, THCTensor_(data)(state, outputT));
  }
  else
  {
 
    #ifdef THC_REAL_IS_FLOAT
    THCusparse_Scsrmm2(state,
    #elif defined(THC_REAL_IS_DOUBLE)
    THCusparse_Dcsrmm2(state,
    #endif
                    transa, transb, nRowWeight, nRowOutput, nColWeight, nnz, alpha, 
                    THCTensor_(data)(state, weightT), THCudaIntTensor_data(state, iRowStartT), 
                    THCudaIntTensor_data(state, colsT), THCTensor_(data)(state, inputT), nColInput, beta, 
                    THCTensor_(data)(state, outputT), nColOutput); 
   }
}

__global__ void linearSparse2DimAccGradParameters(int nnz, real * input, real * gradOut, real * weight, int * rows, int * cols, real scale, int batchSize, int strideGrad, int strideInp)
{
  int j;
  CUDA_KERNEL_LOOP(i, nnz)
  {
     real sum = 0.0;

     //int row = 0;

     //while(rowStart[row+1] <= i)
     //  row++;

     for(j=0; j<batchSize; j++)
     {
	 sum += gradOut[j*strideGrad + rows[i]] * input[j*strideInp + cols[i]];
     }

     weight[i] += sum*scale;
  }
}

__global__ void linearSparse1DimAccGradParameters(int nnz, real * input, real * gradOut, real * weight, int * rows, int * cols, real scale)
{
  CUDA_KERNEL_LOOP(i, nnz)
  {
    // int row = 0;
    // while(i < rowStart[row+1])
    //   row++;
     
     weight[i] += gradOut[rows[i]] * input[cols[i]] * scale;
  }
}

void THNN_(LinearSparse_accGradParameters)(
           THCState * state,
           THCTensor * inputT,
           THCTensor * gradOutT,
           THCTensor * weightT,
           THCudaIntTensor * rowsT,
           THCudaIntTensor * colsT,
           int nnz,
           float scale) {

  THAssert(THCudaTensor_checkGPU(state, 5, inputT, gradOutT, weightT,
                                 rowsT, colsT));

  int nRowInput = inputT->size[0];
  int nColInput = inputT->size[1];
  //int nRowOutput = gradOutT->size[0];
  int nColOutput = gradOutT->size[1];
  long n = nnz;

  real scale_r = (real) scale;

  if(inputT->nDimension == 1)
  {

    linearSparse1DimAccGradParameters<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, THCTensor_(data)(state, inputT), THCTensor_(data)(state, gradOutT), THCTensor_(data)(state, weightT),
        THCudaIntTensor_data(state, rowsT), THCudaIntTensor_data(state, colsT),  scale_r);

  }
  else
  {
      linearSparse2DimAccGradParameters<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, THCTensor_(data)(state, inputT), THCTensor_(data)(state, gradOutT), THCTensor_(data)(state, weightT),
        THCudaIntTensor_data(state, rowsT), THCudaIntTensor_data(state, colsT),  scale_r, nRowInput, nColOutput, nColInput);
   }

  
}

#endif


